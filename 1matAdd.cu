#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <random>

using namespace std;

const int N = 4;

__global__ void matAddKernel(float* d_A, float* d_B, float* d_C, int n) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int fil = blockIdx.y * blockDim.y + threadIdx.y;

	int indice = fil * N + col;


	if (col < N && fil < N) {
		d_C[indice] = d_A[indice] + d_B[indice];
	}
}

__global__ void matAddKernel1(float* d_A, float* d_B, float* d_C, int n) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int fil = blockIdx.y * blockDim.y + threadIdx.y;

	for (int i = 0; i < n; i++) {
		d_C[fil * N+i]= d_A[fil * N + i] + d_B[fil * N + i];
	}
}

__global__ void matAddKernel2(float* d_A, float* d_B, float* d_C, int n) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int fil = blockIdx.y * blockDim.y + threadIdx.y;

	for (int i = 0; i < n; i++) {
		d_C[i * N + col] = d_A[i * N + col] + d_B[i * N + col];
	}
}

void matAdd(float A[][N], float B[][N], float C[][N], int n)
{
	int size = n * n * sizeof(float);
	float* d_A = 0;
	float* d_B = 0;
	float* d_C = 0;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(n, n, 1);


	matAddKernel <<< dimGrid, dimBlock >> > (d_A, d_B, d_C, n);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}


int main() {
	random_device rd;
	mt19937 gen(rd());
	uniform_real_distribution<> dist(1, 100);
	float A[N][N];
	float B[N][N];
	float C[N][N];

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			A[i][j] = dist(gen);
			B[i][j] = dist(gen);
		}
	}

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << A[i][j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << B[i][j] << " ";
		}
		cout << endl;
	}
	cout << endl;


	matAdd(A, B, C, N);


	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << C[i][j] << " ";
		}
		cout << endl;
	}

}
#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <random>

using namespace std;

const int N = 4;

__global__ void matVectMulKernel(float* d_A, float* d_B, float* d_C, int n) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int fil = blockIdx.y * blockDim.y + threadIdx.y;

	float sum = 0;

	for (int j = 0; j < n; j++) {
		sum += d_A[fil * N + j] * d_B[j];
	}
	d_C[fil] = sum;

}


void matVectMul(float A[][N], float B[], float C[], int n)
{
	int sizeM = n * n * sizeof(float);
	int sizeV = n * sizeof(float);
	float* d_A = 0;
	float* d_B = 0;
	float* d_C = 0;

	hipMalloc((void**)&d_A, sizeM);
	hipMalloc((void**)&d_B, sizeV);
	hipMalloc((void**)&d_C, sizeV);

	hipMemcpy(d_A, A, sizeM, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeV, hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(n, n, 1);


	matVectMulKernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C, n);

	hipMemcpy(C, d_C, sizeV, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}


int main() {
	random_device rd;
	mt19937 gen(rd());
	uniform_int_distribution<> dist(1, 10);
	float A[N][N];
	float B[N];
	float C[N];

	for (int i = 0; i < N; i++) {
		B[i] = dist(gen);
		for (int j = 0; j < N; j++) {
			A[i][j] = dist(gen);
		}
	}

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << A[i][j] << " ";
		}
		cout << endl;
	}
	cout << endl;

	for (int i = 0; i < N; i++) {
		cout << B[i] << " ";
	}
	cout << endl;
	cout << endl;


	matVectMul(A, B, C, N);


	for (int i = 0; i < N; i++) {
		cout << C[i] << " ";
	}
	cout << endl;

}